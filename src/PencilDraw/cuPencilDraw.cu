#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include "cuGenStroke.h"
#include "cuToneDraw.h"
#include "cuGenPencil.h"
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

int main(int argc, char** argv)
{
	if (argc != 3)
	{
		cout << "Usage: " << argv[0] << "input" << "pencil" << endl;
		return -1;
	}

	Mat image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
    Mat pencil = imread(argv[2], CV_LOAD_IMAGE_GRAYSCALE);
    Mat S_rst, J_rst, gray_result, color_result;
    Mat fImg;
    image.convertTo(fImg, CV_32FC1);
    
    clock_t start, stop;
    double gs_time, gt_time, gp_time;
    double all_time;
    
    start = clock();
    cuGenStroke(fImg, S_rst, 10, 0.1f);
    cuGenToneMap(image, J_rst);
    cuGenPencil(pencil, J_rst, S_rst, gray_result);
    stop = clock();
    all_time = (double) (stop - start) / CLOCKS_PER_SEC;

    gray_result.convertTo(gray_result, CV_8UC1, 255.0);
    
    imwrite("result/gpu_gray_rst.png", gray_result);

    cout << "Elapsed Time of All: " << all_time << " sec" << endl;
    return 0;
}
