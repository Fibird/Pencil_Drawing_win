#include "hip/hip_runtime.h"
#include <iostream>
#include <ctime>
#include <hip/hip_runtime.h>
#include "cuGenStroke.h"
#include "cuToneDraw.h"
#include "cuGenPencil.h"
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

__global__ void warmup(char *w)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0)
        printf("Warming up ...\n");
}

int main(int argc, char** argv)
{
	if (argc != 3)
	{
		cout << "Usage: " << argv[0] << "input" << "pencil" << endl;
		return -1;
	}

	Mat image = imread(argv[1], CV_LOAD_IMAGE_GRAYSCALE);
    Mat pencil = imread(argv[2], CV_LOAD_IMAGE_GRAYSCALE);
    Mat S_rst, J_rst, gray_result, color_result;
    Mat fImg;
    image.convertTo(fImg, CV_32FC1);
    
    clock_t start, stop;
    double all_time;
    
    // warm up cuda runtime
    char *warmup;
    hipMalloc((void**)&warmup, sizeof(char));

    start = clock();
    cuGenStroke(fImg, S_rst, 10, 0.1f);
    cuGenToneMap(image, J_rst);
    cuGenPencil(pencil, J_rst, S_rst, gray_result);
    stop = clock();
    all_time = (double) (stop - start) / CLOCKS_PER_SEC;

    gray_result.convertTo(gray_result, CV_8UC1, 255.0);
    
    imwrite("result/gpu_gray_rst.png", gray_result);

    cout << "Elapsed Time of All: " << all_time << " sec" << endl;

    return 0;
}
